#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void testkernel(pixel_t * d)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	return;
}

//rows and cols are the row number and col number of the UN-transposed matrix
//rows = h, cols = w
__global__ void transposeDisparity(pixel_t * d_input, pixel_t * d_output, int rows, int cols)
{
	// Allocate appropriate shared memory
	__shared__ float mat[32][32 + 1];

	// Compute input and output index
	int bx = blockIdx.x * blockDim.x;     // Compute block offset - this is number of global threads in X before this block
	int by = blockIdx.y * blockDim.y;     // Compute block offset - this is number of global threads in Y before this block
	int i = bx + threadIdx.x;              // Global input x index - Same as previous kernels
	int j = by + threadIdx.y;              // Global input y index - Same as previous kernels

	int ti = by + threadIdx.x;              // Global output x index - remember the transpose
	int tj = bx + threadIdx.y;              // Global output y index - remember the transpose

	if (i < cols && j < rows)
		mat[threadIdx.y][threadIdx.x] = d_input[j * cols + i];

	__syncthreads();

	// Copy data from shared memory to global memory
	// Check for bounds
	if (ti < rows && tj < cols)
		d_output[tj * rows + rows - 1 - ti] = mat[threadIdx.x][threadIdx.y]; // Switch threadIdx.x and threadIdx.y from input read

}

//rows = m_rows = h, cols = m_cols, c_cols = w
//i: w, j:h
__global__ void columnReduction(pixel_t *d_disparity, pixel_t *d_output,
	int width, int rows, int cols, int c_cols)
{
	__shared__ float mat[32][32 + 1];

	int bx = blockIdx.x * blockDim.x;     // Compute block offset - this is number of global threads in X before this block
	int by = blockIdx.y * blockDim.y;     // Compute block offset - this is number of global threads in Y before this block
	int i = bx + threadIdx.x;              // Global input x index - Same as previous kernels
	int j = by + threadIdx.y;              // Global input y index - Same as previous kernels

	if (i >= cols || j >= rows) return;

	int ti = by + threadIdx.x;              // Global output x index - remember the transpose
	int tj = bx + threadIdx.y;              // Global output y index - remember the transpose

	int i_disparity = i * width;
	int index_disparity = j * cols + i * width;

	//compute mean
	float sum = 0.0f;
	for (int di = 0; di < width; ++di) {
		if (i_disparity + di >= cols) continue;
		sum += d_disparity[index_disparity + di];
	}
	sum /= width;

	mat[threadIdx.y][threadIdx.x] = sum;

	__syncthreads();

	// Copy data from shared memory to global memory
	// Check for bounds
	if (ti < rows && tj < cols) {
		d_output[tj * rows + rows - 1 - ti] = mat[threadIdx.x][threadIdx.y];
	}
		

	//int idx = blockIdx.x * blockDim.x + threadIdx.x;  
	//int jdx = blockIdx.y * blockDim.y + threadIdx.y;

	//if (idx >= rows || jdx >= c_cols) {
	//	return;
	//}

	//int index_in = idx * c_cols + jdx;
	//int index_out = jdx * rows + rows - 1 - idx;

	//int index_dis = idx * cols + jdx * width;
	//int j_original = jdx * width;

	////TODO use median not mean
	//float sum = 0.0f;
	//for (int dj = 0; dj < width; ++dj) {
	//	if (j_original + dj >= cols) continue;
	//	sum += d_disparity[index_dis + dj];
	//}
	//sum /= width;

	//d_output[index_out] = sum;
	////d_output[index_in] = sum;
}

//rows = m_rows = h, x(i) dimension, cols = m_cols, c_cols = w y(j) dimension
__global__ void columnReductionMean(pixel_t * d_disparity, pixel_t * d_output, int width, int rows, int cols, int c_cols)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int jdx = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx >= rows || jdx >= c_cols) {
		return;
	}

	int index_in = idx * c_cols + jdx;

	int index_dis = idx * cols + jdx * width;
	int j_original = jdx * width;

	//TODO use median not mean
	float sum = 0.0f;
	for (int dj = 0; dj < width; ++dj) {
		if (j_original + dj >= cols) continue;
		sum += d_disparity[index_dis + dj];
	}
	sum /= width;

	d_output[index_in] = sum;
}

__global__ void kernComputeGroundDisp(float * d_groundDisp, int h, float baseline, float height, float fu, float v0, float sinTilt, float cosTilt)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= h) return;

	d_groundDisp[h - 1 - idx] = 0.f;
	float val = baseline / height * (fu * sinTilt + (idx - v0) * cosTilt);
	if (val > 0.f) {
		d_groundDisp[h - 1 - idx] = val;
	}

	//groundDisparity[h - 1 - v] = std::max((camera.baseline / camera.height) * (camera.fu * sinTilt + (v - camera.v0) * cosTilt), 0.f);

}

__global__ void kernComputeNegativeLogDataTermGrd(int h, float * d_groundDisparity, float * d_nLogPGaussian_, float * d_fn_, float * d_cquad_, 
	float fv, float tilt, float height, float cf, float sigmaA, float sigmaH, float sigmaD, float dmax, float dmin, float SQRT2, float PI, float pOut, float vhor)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx >= h) return;

	const float tmp = ((vhor - idx) / fv + tilt) / height;
	const float sigmaR2 = cf * cf * (tmp * tmp * sigmaH * sigmaH + sigmaA * sigmaA);
	const float sigma = sqrtf(sigmaD * sigmaD + sigmaR2);

	const float fn = d_groundDisparity[idx];
	const float ANorm = 0.5f * (erff((dmax - fn) / (SQRT2 * sigma)) - erff((dmin - fn) / (SQRT2 * sigma)));
	d_nLogPGaussian_[idx] = logf(ANorm) + logf(sigma * sqrtf(2.f * PI)) - logf(1.f - pOut);
	d_fn_[idx] = fn;

	// coefficient of quadratic part
	d_cquad_[idx] = 1.f / (2.f * sigma * sigma);


	// Gaussian distribution term
	//const int h = static_cast<int>(groundDisparity.size());
	//nLogPGaussian_.resize(h);
	//cquad_.resize(h);
	//fn_.resize(h);
	//for (int v = 0; v < h; v++)
	//{
	//	const float tmp = ((vhor - v) / camera.fv + camera.tilt) / camera.height;
	//	const float sigmaR2 = cf * cf * (tmp * tmp * sigmaH * sigmaH + sigmaA * sigmaA);
	//	const float sigma = sqrtf(sigmaD * sigmaD + sigmaR2);

	//	const float fn = groundDisparity[v];
	//	const float ANorm = 0.5f * (erff((dmax - fn) / (SQRT2 * sigma)) - erff((dmin - fn) / (SQRT2 * sigma)));
	//	nLogPGaussian_[v] = logf(ANorm) + logf(sigma * sqrtf(2.f * PI)) - logf(1.f - pOut);
	//	fn_[v] = fn;

	//	// coefficient of quadratic part
	//	cquad_[v] = 1.f / (2.f * sigma * sigma);
	//}

	/*
	if (d < 0.f)
	return 0.f;

	return std::min(nLogPUniform_, h_nLogPGaussian_[v] + h_cquad_[v] * (d - h_fn_[v]) * (d - h_fn_[v]));
	*/
}

__global__ void kernComputeNegativeLogDataTermObj(int fnmax, float * d_cquad_, float * d_nLogPGaussian_, float fu, float baseline, float sigmaD, float deltaz, float SQRT2, float PI, float pOut, float dmin, float dmax)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx >= fnmax) return;

	const float sigmaZ = idx * idx * deltaz / (fu * baseline);
	const float sigma = sqrtf(sigmaD * sigmaD + sigmaZ * sigmaZ);

	const float ANorm = 0.5f * (erff((dmax - idx) / (SQRT2 * sigma)) - erff((dmin - idx) / (SQRT2 * sigma)));
	d_nLogPGaussian_[idx] = logf(ANorm) + logf(sigma * sqrtf(2.f * PI)) - logf(1.f - pOut);

	d_cquad_[idx] = 1.f / (2.f * sigma * sigma);

	//nLogPGaussian_.resize(fnmax);
	//cquad_.resize(fnmax);
	//for (int fn = 0; fn < fnmax; fn++)
	//{
	//	const float sigmaZ = fn * fn * deltaz / (camera.fu * camera.baseline);
	//	const float sigma = sqrtf(sigmaD * sigmaD + sigmaZ * sigmaZ);

	//	const float ANorm = 0.5f * (erff((dmax - fn) / (SQRT2 * sigma)) - erff((dmin - fn) / (SQRT2 * sigma)));
	//	nLogPGaussian_[fn] = logf(ANorm) + logf(sigma * sqrtf(2.f * PI)) - logf(1.f - pOut);

	//	// coefficient of quadratic part
	//	cquad_[fn] = 1.f / (2.f * sigma * sigma);
	//}
}

__global__ void kernComputeCostsG(int m_w, int m_h, float nLogPUniform_, float * d_costsG, float * d_nLogPGaussian_, float * d_cquad_, float * d_fn_, float * d_disp_colReduced)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int jdx = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx >= m_w || jdx >= m_h) {
		return;
	}
	int pos = idx * m_h + jdx;
	float d = d_disp_colReduced[pos];
	if (d < 0.f) {
		d_costsG[pos] = 0.f;
		return;
	}
	float cost = d_nLogPGaussian_[jdx] + d_cquad_[jdx] * (d - d_fn_[jdx]) * (d - d_fn_[jdx]);
	if (nLogPUniform_ < cost) {
		d_costsG[pos] = nLogPUniform_;
	}
	else {
		d_costsG[pos] = cost;
	}

	/*
	if (d < 0.f)
	return 0.f;

	return std::min(nLogPUniform_, h_nLogPGaussian_[v] + h_cquad_[v] * (d - h_fn_[v]) * (d - h_fn_[v]));
	*/
}

__global__ void kernComputeCostsS(int m_w, int m_h, float * d_costsS, float *d_disp_colReduced,  float nLogPUniform_, float cquad_, float nLogPGaussian_, float fn_)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int jdx = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx >= m_w || jdx >= m_h) {
		return;
	}
	int pos = idx * m_h + jdx;
	float d = d_disp_colReduced[pos];

	if (d < 0.f) {
		d_costsS[pos] = 0.f;
		return;
	}
	float cost = nLogPGaussian_ + cquad_ * (d - fn_) * (d - fn_);
	if (nLogPUniform_ < cost) {
		d_costsS[pos] = nLogPUniform_;
	}
	else {
		d_costsS[pos] = cost;
	}

	/**
	if (d < 0.f)
	return 0.f;

	return std::min(nLogPUniform_, nLogPGaussian_ + cquad_ * (d - fn_) * (d - fn_));
	*/
}

__global__ void kernComputeCostsO(int m_w, int m_h, int fnmax, float * d_costsO, float * d_disp_colReduced, float * d_nLogPGaussian_, float * d_cquad_, float nLogPUniform_)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int jdx = blockIdx.y * blockDim.y + threadIdx.y;
	int zdx = blockIdx.z * blockDim.z + threadIdx.z;

	if (idx >= m_w || jdx >= m_h || zdx >= fnmax) {
		return;
	}

	int pos = idx * m_h + jdx;
	int val_pos = idx * m_h * fnmax + jdx * fnmax + zdx;

	float d = d_disp_colReduced[pos];
	if (d < 0.f) {
		d_costsO[val_pos] = 0.f;
		return;
	}
	float cost = d_nLogPGaussian_[zdx] + d_cquad_[zdx] * (d - zdx) * (d - zdx);
	if (cost > nLogPUniform_) {
		d_costsO[val_pos] = nLogPUniform_;
	}
	else {
		d_costsO[val_pos] = cost;
	}
	/*
	if (d < 0.f)
	return 0.f;

	return std::min(nLogPUniform_, h_nLogPGaussian_[fn] + h_cquad_[fn] * (d - fn) * (d - fn));
	*/
}

__global__ void kernComputeCostsOO(int m_w, int fnmax, int m_h, float * d_costsO, float * d_disp_colReduced, float * d_nLogPGaussian_, float * d_cquad_, float nLogPUniform_)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; //u
	int jdx = blockIdx.y * blockDim.y + threadIdx.y; //fn
	int zdx = blockIdx.z * blockDim.z + threadIdx.z; //v


	if (idx >= m_w || jdx >= fnmax || zdx >= m_h) {
		return;
	}

	int pos = idx * m_h + zdx;
	int val_pos = idx * m_h * fnmax + jdx * m_h + zdx;

	float d = d_disp_colReduced[pos];
	if (d < 0.f) {
		d_costsO[val_pos] = 0.f;
		return;
	}
	float cost = d_nLogPGaussian_[jdx] + d_cquad_[jdx] * (d - jdx) * (d - jdx);
	if (cost > nLogPUniform_) {
		d_costsO[val_pos] = nLogPUniform_;
	}
	else {
		d_costsO[val_pos] = cost;
	}
}

__global__ void kernScanCosts(int m_w, int m_h, float *d_costsG)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= m_w) return;

	thrust::device_ptr<float> thrust_tmpTerm(d_costsG + idx * m_h);
	thrust::device_ptr<float> thrust_costsG(d_costsG + idx * m_h);

	thrust::inclusive_scan(thrust::device, thrust_tmpTerm, thrust_tmpTerm + m_h, thrust_costsG);
}

__global__ void kernScanCostsObj(int m_w, int fnmax, int m_h, float * d_costsO)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; //u
	int jdx = blockIdx.y * blockDim.y + threadIdx.y; //fn

	if (idx >= m_w || jdx >= fnmax) {
		return;
	}

	thrust::device_ptr<float> thrust_tmpTerm(d_costsO + idx * m_h * fnmax + jdx * m_h);
	thrust::device_ptr<float> thrust_costsG(d_costsO + idx * m_h * fnmax + jdx * m_h);

	thrust::inclusive_scan(thrust::device, thrust_tmpTerm, thrust_tmpTerm + m_h, thrust_costsG);
	
}
